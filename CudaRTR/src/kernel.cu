#include "hip/hip_runtime.h"
﻿/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and
 * proprietary rights in and to this software and related documentation.
 * Any use, reproduction, disclosure, or distribution of this software
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA)
 * associated with this source code for terms and conditions that govern
 * your use of this NVIDIA software.
 *
 */

#include <hip/hip_runtime.h>
#include <book.h>
#include <cpu_bitmap.h>
#include <include/hip/hip_runtime_api.h>
#include <thrust/extrema.h>
#include <include/my_utility.h>
#include <include/hittable_list.h>
#include <include/sphere.h>
#include <include/camera.h>

#define DIM 512
 
#define rnd( x ) (x * rand() / RAND_MAX)
#define SAMPLES 100
#define MAX_DEPTH 20

//打印设备信息
void get_device_info() {
    auto device_count = 0;
    hipGetDeviceCount(&device_count);

    if (device_count == 0)
    {
        printf("没有支持CUDA的设备!\n");
        return;
    }
    for (auto dev = 0; dev < device_count; dev++)
    {
        hipSetDevice(dev);
        hipDeviceProp_t device_prop{};
        hipGetDeviceProperties(&device_prop, dev);
        printf("设备 %d: \"%s\"\n", dev, device_prop.name);
        char msg[256];
        sprintf_s(msg, sizeof(msg),
            "global memory大小:        %.0f MBytes "
            "(%llu bytes)\n",
            static_cast<float>(device_prop.totalGlobalMem / 1048576.0f),
            static_cast<unsigned long long>(device_prop.totalGlobalMem));
        printf("%s", msg);
        printf("SM数:                    %2d \n每SM CUDA核心数:           %3d \n总CUDA核心数:             %d \n",
            device_prop.multiProcessorCount,
            _ConvertSMVer2Cores(device_prop.major, device_prop.minor),
            _ConvertSMVer2Cores(device_prop.major, device_prop.minor) *
            device_prop.multiProcessorCount);
        printf("静态内存大小:             %zu bytes\n",
            device_prop.totalConstMem);
        printf("每block共享内存大小:      %zu bytes\n",
            device_prop.sharedMemPerBlock);
        printf("每block寄存器数:          %d\n",
            device_prop.regsPerBlock);
        printf("线程束大小:               %d\n",
            device_prop.warpSize);
        printf("每处理器最大线程数:       %d\n",
            device_prop.maxThreadsPerMultiProcessor);
        printf("每block最大线程数:        %d\n",
            device_prop.maxThreadsPerBlock);
        printf("线程块最大维度大小        (%d, %d, %d)\n",
            device_prop.maxThreadsDim[0], device_prop.maxThreadsDim[1],
            device_prop.maxThreadsDim[2]);
        printf("网格最大维度大小          (%d, %d, %d)\n",
            device_prop.maxGridSize[0], device_prop.maxGridSize[1],
            device_prop.maxGridSize[2]);
        printf("\n");
    }
    printf("************设备信息打印完毕************\n\n");
}

//场景初始化
__device__ double hit_sphere(const point3& center, double radius, const ray& r) {
    vec3 oc = r.origin() - center;
    auto a = dot(r.direction(), r.direction());
    auto half_b = dot(oc, r.direction());
    auto c = oc.length_squared() - radius * radius;
    auto discriminant = half_b * half_b - a * c;
    if (discriminant < 0)   return -1.0;
    else return (-half_b - sqrt(discriminant)) /  a;
}

//计算光线颜色
__device__ color ray_color(ray& r, hittable_list*const dev_world, hiprandStateXORWOW_t* state) {
    hit_record rec;
    size_t depth = MAX_DEPTH;
    color result = color(1.0, 1.0, 1.0);
    ray temp;
    color attenuation;

    while (true) {
        if (depth <= 0) return color(0.0, 0.0, 0.0);
        if (!dev_world->hit(r, 0.001, infinity, rec)) {
            vec3 unit_direction = unit_vector(r.direction());
            auto t = 0.5 * (unit_direction.y() + 1.0);
            return result * ((1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0));
        }
        depth--;
        if (!rec.mat_ptr->scatter(r, rec, attenuation, temp, state)) return color(0.0, 0.0, 0.0);
        r = temp;
        result = result * attenuation;
    }
    return result;
    //while ((depth > 0) && dev_world->hit(r, 0.001, infinity, rec)) {
    //    depth--;
    //    if (rec.mat_ptr->scatter(r, rec, attenuation, temp, state)) {
    //        r = temp;
    //        result = cross(result, attenuation);
    //    }
    //    else {
    //        return color(0.0, 0.0, 0.0);
    //    }
    //}
    //if (depth == 0) return color(0.0, 0.0, 0.0);
    //vec3 unit_direction = unit_vector(r.direction());
    //auto t = 0.5 * (unit_direction.y() + 1.0);
    //return cross(result, (1.0 - t) * color(1.0, 1.0, 1.0) + t * color(0.5, 0.7, 1.0));
}

//GPU计算像素
__global__ void kernel(unsigned char* ptr, Moon::camera** cam, hittable_list** world) {
    
    // map from threadIdx/BlockIdx to pixel position
     int x = threadIdx.x + blockIdx.x * blockDim.x;
     int y = threadIdx.y + blockIdx.y * blockDim.y;
     int offset = x + y * blockDim.x * gridDim.x;
     int seed = offset;
     hiprandStateXORWOW_t rand_state;
     hiprand_init(seed, 0, 0, &rand_state);
     
     color pixel_color(0.0, 0.0, 0.0);
     for (size_t s = 0; s < SAMPLES; ++s) {
         double u = double(x + Moon::random_double(&rand_state)) / (double)(DIM - 1);
         double v = double(y + Moon::random_double(&rand_state)) / (double)(DIM - 1);
         ray r = (*cam)->get_ray(u, v);

         pixel_color += ray_color(r, *world, &rand_state);
     }

     double r = pixel_color.x();
     double g = pixel_color.y();
     double b = pixel_color.z();
     double scale = 1.0 / (double)SAMPLES;
     r = sqrt(scale * r);
     g = sqrt(scale * g);
     b = sqrt(scale * b);
     ptr[offset * 4 + 0] = (int)(Moon::clamp(r, 0.0, 0.999) * 255);
     ptr[offset * 4 + 1] = (int)(Moon::clamp(g, 0.0, 0.999) * 255);
     ptr[offset * 4 + 2] = (int)(Moon::clamp(b, 0.0, 0.999) * 255);
     ptr[offset * 4 + 3] = 255;
 }

 // globals needed by the update routine
struct DataBlock {
    unsigned char* dev_bitmap;
};
// 如果HANDLE_ERROR有问题的请参考下文"mydef.h"
int main(void) {

    get_device_info();

    //图片大小
    const auto aspect_ratio = 1.0;
    const int image_width = 512;
    const int image_height = static_cast<int>(image_width / aspect_ratio);

    DataBlock   data;   //gpu数据块
    // 计时器
    hipEvent_t     start, stop;
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));
    HANDLE_ERROR(hipEventRecord(start, 0));

    CPUBitmap bitmap(DIM, DIM, &data);  //cpu图像
    unsigned char* dev_bitmap;

    // allocate memory on the GPU for the output bitmap
    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap,
        bitmap.image_size())); 

    //相机加载
    Moon::camera** device_cam = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&device_cam, sizeof(Moon::camera**)));
    Moon::make_camera_device << <1, 1 >> > (device_cam);
    hipDeviceSynchronize();
    hipGetLastError();
    
    //场景加载
    hittable_list** device_world = nullptr;
    HANDLE_ERROR(hipMalloc((void**)&device_world, sizeof(hittable_list**)));
    make_scene_device << <1, 1 >> > (device_world, 2);
    hipDeviceSynchronize();
    hipGetLastError();

    // generate a bitmap from our sphere data
    dim3    grids(DIM / 16, DIM / 16);
    dim3    threads(16, 16);
    hipDeviceSynchronize();
    kernel << <grids, threads >> > (dev_bitmap, device_cam, device_world);
    hipDeviceSynchronize();

    // copy our bitmap back from the GPU for display
    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
        bitmap.image_size(),
        hipMemcpyDeviceToHost));

    // get stop time, and display the timing results
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));
    float   elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
        start, stop));
    printf("Time to generate:  %3.1f ms\n", elapsedTime);
    printf("FPS: %3.1f\n", 1000.0 / elapsedTime);

    HANDLE_ERROR(hipFree(device_world));
    destroy_scene_device << <1, 1 >> > (device_world);
    HANDLE_ERROR(hipFree(device_cam));
    destroy_camera_device << <1, 1 >> > (device_cam);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    HANDLE_ERROR(hipFree(dev_bitmap));

    // display
    bitmap.display_and_exit();
}


//#define DIM 1024
//
//#define rnd( x ) (x * rand() / RAND_MAX)
//#define INF     2e10f
//
//struct Sphere {
//    float   r, b, g;
//    float   radius;
//    float   x, y, z;
//    __device__ float hit(float ox, float oy, float* n) {
//        float dx = ox - x;
//        float dy = oy - y;
//        if (dx * dx + dy * dy < radius * radius) {
//            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
//            *n = dz / sqrtf(radius * radius);
//            return dz + z;
//        }
//        return -INF;
//    }
//};
//#define SPHERES 20
//
//__constant__ Sphere s[SPHERES];
//
//__global__ void kernel(unsigned char* ptr) {
//    // map from threadIdx/BlockIdx to pixel position
//    int x = threadIdx.x + blockIdx.x * blockDim.x;
//    int y = threadIdx.y + blockIdx.y * blockDim.y;
//    int offset = x + y * blockDim.x * gridDim.x;
//    float   ox = (x - DIM / 2);// move to center
//    float   oy = (y - DIM / 2);
//
//    float   r = 0, g = 0, b = 0;
//    float   maxz = -INF;
//    for (int i = 0; i < SPHERES; i++) {
//        float   n;
//        float   t = s[i].hit(ox, oy, &n);
//        if (t > maxz) {
//            float fscale = n;
//            r = s[i].r * fscale;
//            g = s[i].g * fscale;
//            b = s[i].b * fscale;
//            maxz = t;
//        }
//    }
//
//    ptr[offset * 4 + 0] = (int)(r * 255);
//    ptr[offset * 4 + 1] = (int)(g * 255);
//    ptr[offset * 4 + 2] = (int)(b * 255);
//    ptr[offset * 4 + 3] = 255;
//}
//
//// globals needed by the update routine
//struct DataBlock {
//    unsigned char* dev_bitmap;
//};
//// 如果HANDLE_ERROR有问题的请参考下文"mydef.h"
//int main(void) {
//    DataBlock   data;
//    // capture the start time
//    hipEvent_t     start, stop;
//    HANDLE_ERROR(hipEventCreate(&start));
//    HANDLE_ERROR(hipEventCreate(&stop));
//    HANDLE_ERROR(hipEventRecord(start, 0));
//
//    CPUBitmap bitmap(DIM, DIM, &data);
//    unsigned char* dev_bitmap;
//
//    // allocate memory on the GPU for the output bitmap
//    HANDLE_ERROR(hipMalloc((void**)&dev_bitmap,
//        bitmap.image_size()));
//
//    // allocate temp memory, initialize it, copy to constant
//    // memory on the GPU, then free our temp memory
//    Sphere* temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
//    for (int i = 0; i < SPHERES; i++) {
//        temp_s[i].r = rnd(1.0f);
//        temp_s[i].g = rnd(1.0f);
//        temp_s[i].b = rnd(1.0f);
//        temp_s[i].x = rnd(1000.0f) - 500;
//        temp_s[i].y = rnd(1000.0f) - 500;
//        temp_s[i].z = rnd(1000.0f) - 500;
//        temp_s[i].radius = rnd(100.0f) + 20;
//    }
//    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(s), temp_s,
//        sizeof(Sphere) * SPHERES));
//    free(temp_s);
//
//    // generate a bitmap from our sphere data
//    dim3    grids(DIM / 16, DIM / 16);
//    dim3    threads(16, 16);
//    kernel << <grids, threads >> > (dev_bitmap);
//
//    // copy our bitmap back from the GPU for display
//    HANDLE_ERROR(hipMemcpy(bitmap.get_ptr(), dev_bitmap,
//        bitmap.image_size(),
//        hipMemcpyDeviceToHost));
//
//    // get stop time, and display the timing results
//    HANDLE_ERROR(hipEventRecord(stop, 0));
//    HANDLE_ERROR(hipEventSynchronize(stop));
//    float   elapsedTime;
//    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime,
//        start, stop));
//    printf("Time to generate:  %3.1f ms\n", elapsedTime);
//
//    HANDLE_ERROR(hipEventDestroy(start));
//    HANDLE_ERROR(hipEventDestroy(stop));
//
//    HANDLE_ERROR(hipFree(dev_bitmap));
//
//    // display
//    bitmap.display_and_exit();
//}