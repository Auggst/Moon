#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <thrust/random.h>

hipError_t addWithCuda(double*c, const double*a, const double*b, unsigned int size);

__global__ void addKernel(double *c, const double*a, const double*b)
{
    int i = threadIdx.x;
    thrust::minstd_rand rng;
    rng.seed(i);
    thrust::uniform_real_distribution<double> dist(-1.0, 1.0);
    c[i] = dist(rng) + b[i];
}

int main()
{
    const int arraySize = 5;
    const double a[arraySize] = { 1.0, 2.0, 3.0, 4.0, 5.0 };
    const double b[arraySize] = { 10.0, 20.0, 30.0, 40.0, 50.0 };
    double c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(double *c, const double*a, const double*b, unsigned int size)
{
    double*dev_a = 0;
    double*dev_b = 0;
    double*dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
